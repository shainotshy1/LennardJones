#include "hip/hip_runtime.h"
#include "molecule_sim_helper.cuh"
#include "cuda_memory_utils.cuh"
#include "helper_utils.cuh"

__global__ void simulate_molecules_gpu(int* env_molecules,
	double* pos_x,
	double* pos_y,
	double* vel_x,
	double* vel_y,
	double* radii,
	int num_molecules,
	int num_environments,
	double* env_pos_x,
	double* env_pos_y,
	double* env_dim_x,
	double* env_dim_y,
	int* grid,
	int cell_dim_x,
	int cell_dim_y,
	double global_dim_x,
	double global_dim_y,
	double global_pos_x,
	double global_pos_y,
	double grid_dim)
{
	//To Do: Implement Lennard Jones
	//To Do: Parrallelize molecules of all environments into ONE kernel call:
		//Contains an array of length n where n = # of environments
		//Array has number of molecules in each environment so we know what segments of the pos/vel/radii vectors map to which environments

	int i = blockIdx.x * blockDim.x + threadIdx.x; //global molecule index

	if (i >= num_molecules) return;

	int j = 0; //environment index of molecule
	int k = 0;
	while (i >= k + env_molecules[j]) {
		k += env_molecules[j];
		j++;
	}
	
	simulate_molecules(i,
		pos_x,
		pos_y,
		vel_x,
		vel_y,
		radii,
		grid,
		cell_dim_x,
		cell_dim_y,
		global_dim_x,
		global_dim_y,
		global_pos_x,
		global_pos_y,
		grid_dim);

	bound_molecules(i,
		pos_x,
		pos_y,
		vel_x,
		vel_y,
		radii,
		env_pos_x[j],
		env_pos_y[j],
		env_dim_x[j],
		env_dim_y[j]);
	
	update_molecule(i, 
		pos_x, 
		pos_y, 
		vel_x, 
		vel_y,
		grid,
		global_pos_x,
		global_pos_y,
		grid_dim,
		cell_dim_x,
		cell_dim_y);
}

//TODO: Generalize for multiple environments; fix nested for loop to shift bounds of surrounding molecules to depend on environments
__device__ void simulate_molecules(int mol_index,
	double* pos_x,
	double* pos_y,
	double* vel_x,
	double* vel_y,
	double* radii,
	int* grid,
	int cell_dim_x,
	int cell_dim_y,
	double global_dim_x,
	double global_dim_y,
	double global_pos_x,
	double global_pos_y,
	double grid_dim)
{
#if 1

	int row = (pos_x[mol_index] - global_pos_x) / grid_dim;
	int col = (pos_y[mol_index] - global_pos_y) / grid_dim;
	int grid_index = (int)(row * cell_dim_x + col);

	if (grid_index >= cell_dim_x * cell_dim_y) {
		printf("ERROR index out of bound molecule");
	}
#else
	int index = find_grid_index(pos_x[i],
		pos_y[i],
		global_pos_x,
		global_pos_y,
		cell_dim_x,
		cell_dim_y,
		grid_dim);
#endif
	if (grid[grid_index] != mol_index) {
		printf("Error %d VS %d\n\n", grid[grid_index], mol_index);
	}

	int max_multiplier = 4;
	int cells = cell_dim_x * cell_dim_y;
	for (int i = grid_index - max_multiplier / 2; i < grid_index + (max_multiplier + 1)/ 2; i++) {
		if (i % cell_dim_x == 0 && (grid_index + 1) % cell_dim_x == 0) { //check if molecule is on the right edge of the environment
			continue;
		}
		if ((i + 1) % cell_dim_x == 0 && grid_index % cell_dim_x == 0) {
			continue;
		}
		
		for (int j = -max_multiplier/2; j < (max_multiplier + 1)/2; j++) {
			if (i == grid_index) {
				continue;
			}

			int other_mol_index = j * cell_dim_x + i;
			if (other_mol_index > 0 && other_mol_index < cells) {
				simulate_interaction(mol_index, 
					other_mol_index,
					pos_x,
					pos_y,
					radii);
			}
		}
	}
}

__device__ void simulate_interaction(int mol_index,
	int other_mol_index,
	double* pos_x,
	double* pos_y,
	double* radii)
{
	double x1 = pos_x[mol_index];
	double y1 = pos_y[mol_index];
	double r1 = radii[mol_index];

	double x2 = pos_x[other_mol_index];
	double y2 = pos_y[other_mol_index];
	double r2 = radii[mol_index];

	double sqr_dist = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1);
	double sqr_rad = (r1 + r2) * (r1 + r2);
	if (sqr_dist <= sqr_rad) {
		printf("Collision!\n");
	}
}

__device__ void bound_molecules(int i,
	double* pos_x,
	double* pos_y,
	double* vel_x,
	double* vel_y,
	double* radii,
	double env_pos_x,
	double env_pos_y,
	double env_dim_x,
	double env_dim_y)
{
	double dif_pos_x = env_pos_x + env_dim_x - pos_x[i];
	double dif_pos_y = env_pos_y + env_dim_y - pos_y[i];
	double radius = radii[i];
	if (dif_pos_x >= env_dim_x - radius || dif_pos_x <= radius) {
		vel_x[i] *= -1;
	}
	if (dif_pos_y >= env_dim_y - radius || dif_pos_y <= radius) {
		vel_y[i] *= -1;
	}
}

__device__ void update_molecule(int i, 
	double* pos_x, 
	double* pos_y, 
	double* vel_x, 
	double* vel_y,
	int* grid,
	double global_pos_x,
	double global_pos_y,
	double grid_dim,
	double cell_dim_x,
	double cell_dim_y)
{
	pos_x[i] += vel_x[i];
	pos_y[i] += vel_y[i];

	int row = (pos_x[i] - global_pos_x) / grid_dim;
	int col = (pos_y[i] - global_pos_y) / grid_dim;
	int index = (int)(row * cell_dim_x + col);

	if (index >= cell_dim_x * cell_dim_y) {
		printf("ERROR index out of bound molecule");
	}

	grid[index] = i;
}